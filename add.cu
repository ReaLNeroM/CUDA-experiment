
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cassert>
#include <cstdlib>

__global__ void add_1(int n, double *x, double *y, double *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    for(int i = tid; i < n; i += stride){
        ans[i] = x[i] + y[i];
    }
}

__global__ void add_2(int n, double *x, double *y, double *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    for(int i = tid; i < n; i += stride){
        ans[i] = x[i] + y[i];
        __syncthreads();
    }
}


__global__ void add_3(int n, double *x, double *y, double *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    int rowstart = ((long long) tid * n) / stride;
    int rowend = ((long long) (tid + 1) * n) / stride;

    for(int i = rowstart; i < rowend; i++){
        ans[i] = x[i] + y[i];
    }
}

__global__ void add_4(int n, double *x, double *y, double *ans){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    ans[i] = x[i] + y[i];
}

int main(int argc, char** argv){
    assert(argc == 2);
    int algo_type = std::stoi(argv[1]);

    int n = (1 << 25);

    double *x, *y, *ans;

    x = (double *) malloc(n * sizeof(double));
    assert(x != NULL);
    y = (double *) malloc(n * sizeof(double));
    assert(y != NULL);
    ans = (double *) malloc(n * sizeof(double));
    assert(ans != NULL);

    for(int i = 0; i < n; i++){
        x[i] = 1.0;
        y[i] = 2.0;
    }

    double *d_x, *d_y, *d_ans;
    hipMalloc(&d_x, n * sizeof(double));
    assert(d_x != NULL);
    hipMalloc(&d_y, n * sizeof(double));
    assert(d_y != NULL);
    hipMalloc(&d_ans, n * sizeof(double));
    assert(d_ans != NULL);

    hipMemcpy(d_x, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (n + 256 - 1) / 256;

    auto start = std::chrono::high_resolution_clock::now();
    auto finish = std::chrono::high_resolution_clock::now();

    start = std::chrono::high_resolution_clock::now();
    if(algo_type == 1){
        add_1<<<1, 256>>>(n, d_x, d_y, d_ans);
    } else if(algo_type == 2){
        add_2<<<1, 256>>>(n, d_x, d_y, d_ans);
    } else if(algo_type == 3){
        add_3<<<1, 256>>>(n, d_x, d_y, d_ans);
    } else if(algo_type == 4){
        add_4<<<grid_size, 256>>>(n, d_x, d_y, d_ans);
    }
    hipDeviceSynchronize();
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "add_" << algo_type << ": " << std::chrono::duration_cast<std::chrono::milliseconds>(finish-start).count() << "ms\n";

    hipMemcpy(ans, d_ans, n * sizeof(double), hipMemcpyDeviceToHost);

    double err = 0.0;
    for(int i = 0; i < n; i++){
        err += abs(ans[i] - 3.0);
    }

    std::cout << err << '\n';

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_ans);

    free(x);
    free(y);
    free(ans);
}