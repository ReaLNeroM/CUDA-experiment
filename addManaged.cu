
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cassert>

__global__ void add_1(int n, float *x, float *y, float *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    for(int i = tid; i < n; i += stride){
        ans[i] = x[i] + y[i];
    }
}

__global__ void add_2(int n, float *x, float *y, float *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    for(int i = tid; i < n; i += stride){
        ans[i] = x[i] + y[i];
        __syncthreads();
    }
}


__global__ void add_3(int n, float *x, float *y, float *ans){
    int tid = threadIdx.x;
    int stride = blockDim.x;

    int rowstart = ((long long) tid * n) / stride;
    int rowend = ((long long) (tid + 1) * n) / stride;

    for(int i = rowstart; i < rowend; i++){
        ans[i] = x[i] + y[i];
    }
}

__global__ void add_4(int n, float *x, float *y, float *ans){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < n){
        ans[i] = x[i] + y[i];
    }
}

int main(int argc, char** argv){
    assert(argc == 2);
    int algo_type = std::stoi(argv[1]);

    int n = (1 << 25);

    float *x, *y, *ans;

    hipMallocManaged(&x, n * sizeof(float));
    assert(x != NULL);
    hipMallocManaged(&y, n * sizeof(float));
    assert(y != NULL);
    hipMallocManaged(&ans, n * sizeof(float));
    assert(ans != NULL);

    for(int i = 0; i < n; i++){
        x[i] = 1.0;
        y[i] = 2.0;
    }

    int grid_size = (n + 256 - 1) / 256;

    auto start = std::chrono::high_resolution_clock::now();
    auto finish = std::chrono::high_resolution_clock::now();

    start = std::chrono::high_resolution_clock::now();
    if(algo_type == 1){
        add_1<<<1, 256>>>(n, x, y, ans);
    } else if(algo_type == 2){
        add_2<<<1, 256>>>(n, x, y, ans);
    } else if(algo_type == 3){
        add_3<<<1, 256>>>(n, x, y, ans);
    } else if(algo_type == 4){
        add_4<<<grid_size, 256>>>(n, x, y, ans);
    }

    hipDeviceSynchronize();
    finish = std::chrono::high_resolution_clock::now();
    std::cout << "add_" << algo_type << ": " << std::chrono::duration_cast<std::chrono::milliseconds>(finish-start).count() << "ms\n";

    float err = 0.0;
    for(int i = 0; i < n; i++){
        err += abs(ans[i] - 3.0);
    }

    std::cout << err << '\n';

    hipFree(x);
    hipFree(y);
    hipFree(ans);
}